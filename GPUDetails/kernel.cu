﻿#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int deviceCount;

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable GPUs detected." << std::endl;
        return 1;
    }

    for (int i = 0; i < deviceCount; i++) {
        hipGetDeviceProperties(&prop, i);

        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Shared Memory per Block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Registers per Block: " << prop.regsPerBlock << std::endl;
        std::cout << "  Warp Size: " << prop.warpSize << std::endl;
        std::cout << "  Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Block Dimensions: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")" << std::endl;
        std::cout << "  Max Grid Dimensions: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << std::endl;
    }

    return 0;
}